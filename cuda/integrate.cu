
#include <hip/hip_runtime.h>
////
//// Created by Renhui on 2023/1/7.
////
//#include <cuda_runtime.h>
//#include <cuda.h>
//
//
//#include "camera.h"
//#include "scene.h"
//#include "Construct.h"
////#include "interaction.h"
//
//class Integrate {
//public:
//    Integrate(cuda::shared_ptr<Camera> cam,
//               cuda::shared_ptr<Scene> scene, int spp,Grids_data &gridsData,  float iso_value, float var=0.001, float step_scale=1){
//
//    }
//
//    void render() const{
//
//    }
//
//    [[nodiscard]] float opacity_transfer(float value) const{
//
//    }
//
//    [[nodiscard]] static Vec3f color_transfer(float val) {
//
//    }
//
//    [[nodiscard]] float interpolation(Vec3f pos, uint32_t grid_idx_bm) const{
//
//    }
//
//    static float opacity_correction(float actual_step, float step_base, float opacity){
//
//    }
//
//    Vec3f front_to_back(Ray &ray) const{
//
//    }
//
//    [[nodiscard]] float step_Base(Vec3f pos, uint32_t grid_idx_bm) const{
//
//    }
//
//private:
//    cuda::shared_ptr<Camera> camera;
//    cuda::shared_ptr<Scene> scene;
////    int max_depth;
//    int spp;
////    float dist_limit;
//    Grids_data &gridsData;
//    Kdtree kdtree;
//    float variance;
//    float iso_value;
//    float step_scale;
//};